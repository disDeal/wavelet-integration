#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <random>
#include <vector>

auto randomNumberBetween = [](int low, int high) {
  auto randomFunc =
      [distribution_ = std::uniform_int_distribution<int>(low, high),
       random_engine_ = std::mt19937{std::random_device{}()}]() mutable {
        return distribution_(random_engine_);
      };
  return randomFunc;
};

__global__ void wavelet(float* a, float* b, int N) {
  __shared__ float arr1[1024];
  __shared__ float arr1[1024];
  float* p1 = arr1;
  float* p2 = arr2;
  float* p3;

  int idx = threadIdx.x;
  arr1[i] = a[i]

  __syncthreads();

  int off = N / 2;

  int k1 = i % 2;
  int k = k1 * (-2) + 1;
  arr2[i / 2 + off * k1] = arr1[i / 2] + arr1[i / 2 + 1] * (float)k;

  while (off > 0) {
    p3 = p1;
    p1 = p2;
    p2 = p3;
    off /= 2;
    p2[i / 2 + off * k1] = p1[i / 2] + p1[i / 2 + 1] * (float)k;
    __syncthreads();
  }
  b[i] = p2[i];
}

void print(float* arr, int N) {
  for (size_t i = 0; i < N; i++) {
    std::cout << arr[i] << ' ';
  }
  std::cout << std::endl;
}

int main() {
  int N = 8;
  // std::vector<float> numbers;
  // std::generate_n(std::back_inserter(numbers), N,
  //                 randomNumberBetween(0., 10.));
  float numbers[] = {1., 2., 3., 4., 5., 6., 7., 8.};
  float wave[N];

  print(numbers, N);

  wavelet(numbers, wave, N);

  print(wave, N);

  return 0;
}
